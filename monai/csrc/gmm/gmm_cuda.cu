
#include <hip/hip_runtime.h>
/*
Copyright 2020 - 2021 MONAI Consortium
Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at
    http://www.apache.org/licenses/LICENSE-2.0
Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/

#define MIXTURES 2

__device__ __forceinline__ float get_component(uchar4 pixel, int i)
{
    switch (i)
    {
        case 0 :
            return 1.0f;

        case 1 :
            return pixel.x;

        case 2 :
            return pixel.y;

        case 3 :
            return pixel.z;

        case 4 :
            return pixel.x * pixel.x;

        case 5 :
            return pixel.x * pixel.y;

        case 6 :
            return pixel.x * pixel.z;

        case 7 :
            return pixel.y * pixel.y;

        case 8 :
            return pixel.y * pixel.z;

        case 9 :
            return pixel.z * pixel.z;
    };

    return 0.0f;
}

__device__ __forceinline__ float get_constant(float *gmm, int i)
{
    const float epsilon = 1.0e-3f;

    switch (i)
    {
        case 0 :
            return 0.0f;

        case 1 :
            return 0.0f;

        case 2 :
            return 0.0f;

        case 3 :
            return 0.0f;

        case 4 :
            return gmm[1] * gmm[1] + epsilon;

        case 5 :
            return gmm[1] * gmm[2];

        case 6 :
            return gmm[1] * gmm[3];

        case 7 :
            return gmm[2] * gmm[2] + epsilon;

        case 8 :
            return gmm[2] * gmm[3];

        case 9 :
            return gmm[3] * gmm[3] + epsilon;
    };

    return 0.0f;
}


// Tile Size: 32x32, Block Size 32xwarp_N
template<int warp_N, bool create_gmm_flags>
__global__ void GMMReductionKernel(int gmm_idx, float *gmm, int gmm_pitch, const uchar4 *image, char *alpha, int width, int height, unsigned int *tile_gmms)
{
    __shared__ uchar4 s_lists[32 * 32];
    __shared__ volatile float s_gmm[32 * warp_N];
    __shared__ float s_final[warp_N];

    __shared__ int gmm_flags[32];

    const int warp_idx = threadIdx.y;
    const int thread_idx = threadIdx.y * 32 + threadIdx.x;
    const int lane_idx = threadIdx.x;

    float *block_gmm = &gmm[(gridDim.x * gridDim.y * gmm_idx + blockIdx.y * gridDim.x + blockIdx.x) * gmm_pitch];
    volatile float *warp_gmm = &s_gmm[warp_idx * 32];

    if (create_gmm_flags)
    {
        if (threadIdx.y == 0)
        {
            gmm_flags[threadIdx.x] = 0;
        }

        __syncthreads();
    }
    else
    {
        unsigned int gmm_mask = tile_gmms[blockIdx.y * gridDim.x + blockIdx.x];

        if ((gmm_mask & (1u << gmm_idx)) == 0)
        {

            if (threadIdx.x < 10 && threadIdx.y ==0)
            {
                block_gmm[threadIdx.x] = 0.0f;
            }

            return;
        }
    }

    int list_idx = 0;

    int y = blockIdx.y * 32 + threadIdx.y;
    int x = blockIdx.x * 32 + threadIdx.x;

    // Build lists of pixels that belong to this GMM

    for (int k=0; k < (32/warp_N); ++k)
    {
        if (x < width && y < height)
        {
            int my_gmm_idx = alpha[y * width + x];

            if (my_gmm_idx != -1)
            {
                if (create_gmm_flags)
                {
                    gmm_flags[my_gmm_idx] = 1;
                }
    
                if (my_gmm_idx == gmm_idx)
                {
                    uchar4 pixel = image[y * width + x];
                    s_lists[thread_idx + list_idx * (32*warp_N)] = pixel;
                    ++list_idx;
                }
            }
        }

        y += warp_N;
    }

    __syncthreads();

    if (threadIdx.y == 0 && create_gmm_flags)
    {
        tile_gmms[blockIdx.y * gridDim.x + blockIdx.x] = __ballot_sync(0xFFFFFFFF, gmm_flags[threadIdx.x] > 0);
    }

    // Reduce for each global GMM element

    for (int i=0; i<10; ++i)
    {
        float thread_gmm;

        if (i == 0)
        {
            // thread_gmm = list_idx for first component
            thread_gmm = list_idx;
        }
        else
        {
            thread_gmm = list_idx > 0 ? get_component(s_lists[thread_idx],i) : 0.0f;

            for (int k=1; k<(32/warp_N) && k < list_idx; ++k)
            {
                thread_gmm += get_component(s_lists[thread_idx + k * (32*warp_N)], i);
            }
        }

        warp_gmm[lane_idx] = thread_gmm;

        // Warp Reductions
        thread_gmm += warp_gmm[(lane_idx + 16) & 31];
        warp_gmm[lane_idx] = thread_gmm;

        thread_gmm += warp_gmm[(lane_idx + 8) & 31];
        warp_gmm[lane_idx] = thread_gmm;

        thread_gmm += warp_gmm[(lane_idx + 4) & 31];
        warp_gmm[lane_idx] = thread_gmm;

        thread_gmm += warp_gmm[(lane_idx + 2) & 31];
        warp_gmm[lane_idx] = thread_gmm;

        thread_gmm += warp_gmm[(lane_idx + 1) & 31];
        s_final[warp_idx] = thread_gmm;

        __syncthreads();

        // Final Reduction
        if (warp_idx ==0 && lane_idx == 0)
        {
            for (int j=1; j<warp_N; ++j)
            {
                thread_gmm += s_final[j];
            }

            block_gmm[i] = thread_gmm;
        }
    }
}

__constant__ int det_indices[] = { (9 << (4*4)) + (4 << (3*4)) + (6 << (2*4)) + (5 << (1*4)) + (4 << (0*4)),
                                   (5 << (4*4)) + (8 << (3*4)) + (6 << (2*4)) + (6 << (1*4)) + (7 << (0*4)),
                                   (5 << (4*4)) + (8 << (3*4)) + (7 << (2*4)) + (8 << (1*4)) + (9 << (0*4))
                                 };

__constant__ int inv_indices[] = { (4 << (5*4)) + (5 << (4*4)) + (4 << (3*4)) + (5 << (2*4)) + (6 << (1*4)) + (7 << (0*4)),
                                   (7 << (5*4)) + (6 << (4*4)) + (9 << (3*4)) + (8 << (2*4)) + (8 << (1*4)) + (9 << (0*4)),
                                   (5 << (5*4)) + (4 << (4*4)) + (6 << (3*4)) + (6 << (2*4)) + (5 << (1*4)) + (8 << (0*4)),
                                   (5 << (5*4)) + (8 << (4*4)) + (6 << (3*4)) + (7 << (2*4)) + (9 << (1*4)) + (8 << (0*4))
                                 };


// One block per GMM, 32*warp_N threads (1-dim)
template <int warp_N, bool invertSigma>
__global__ void GMMFinalizeKernel(float *gmm, float *gmm_scratch, int gmm_pitch, int N)
{
    __shared__ volatile float s_gmm[warp_N*32];
    __shared__ float s_final[warp_N];
    __shared__ float final_gmm[15];

    const int thread_N = warp_N * 32;

    float *gmm_partial = &gmm_scratch[N*blockIdx.x*gmm_pitch];

    volatile float *warp_gmm = &s_gmm[threadIdx.x & 0x0ffe0];

    int thread_idx = threadIdx.x;
    int lane_idx = threadIdx.x & 31;
    int warp_idx = threadIdx.x >> 5;

    float norm_factor = 1.0f;

    for (int i=0; i<10; ++i)
    {
        float thread_gmm = 0.0f;

        for (int j=thread_idx; j < N; j+= thread_N)
        {
            thread_gmm += gmm_partial[j * gmm_pitch + i];
        }

        warp_gmm[lane_idx] = thread_gmm;

        // Warp Reduction
        thread_gmm += warp_gmm[(lane_idx + 16) & 31];
        warp_gmm[lane_idx] = thread_gmm;

        thread_gmm += warp_gmm[(lane_idx + 8) & 31];
        warp_gmm[lane_idx] = thread_gmm;

        thread_gmm += warp_gmm[(lane_idx + 4) & 31];
        warp_gmm[lane_idx] = thread_gmm;

        thread_gmm += warp_gmm[(lane_idx + 2) & 31];
        warp_gmm[lane_idx] = thread_gmm;

        thread_gmm += warp_gmm[(lane_idx + 1) & 31];

        s_final[warp_idx] = thread_gmm;

        __syncthreads();

        // Final Reduction
        if (warp_idx ==0 && lane_idx == 0)
        {
            for (int j=1; j<warp_N; ++j)
            {
                thread_gmm += s_final[j];
            }

            final_gmm[i] = norm_factor * thread_gmm - get_constant(final_gmm, i);

            if (i == 0)
            {
                if (thread_gmm > 0)
                {
                    norm_factor = 1.0f / thread_gmm;
                }
            }
        }
    }

    if (threadIdx.y == 0)
    {
        // Compute det(Sigma) using final_gmm [10-14] as scratch mem

        if (threadIdx.x < 5)
        {

            int idx0 = (det_indices[0] & (15 << (threadIdx.x * 4))) >> (threadIdx.x * 4);
            int idx1 = (det_indices[1] & (15 << (threadIdx.x * 4))) >> (threadIdx.x * 4);
            int idx2 = (det_indices[2] & (15 << (threadIdx.x * 4))) >> (threadIdx.x * 4);

            final_gmm[10 + threadIdx.x] = final_gmm[idx0] * final_gmm[idx1] * final_gmm[idx2];

            float det = final_gmm[10] + 2.0f * final_gmm[11] - final_gmm[12] - final_gmm[13] - final_gmm[14];
            final_gmm[10] = det;
        }

        // Compute inv(Sigma)
        if (invertSigma && threadIdx.x < 6)
        {
            int idx0 = (inv_indices[0] & (15 << (threadIdx.x * 4))) >> (threadIdx.x * 4);
            int idx1 = (inv_indices[1] & (15 << (threadIdx.x * 4))) >> (threadIdx.x * 4);
            int idx2 = (inv_indices[2] & (15 << (threadIdx.x * 4))) >> (threadIdx.x * 4);
            int idx3 = (inv_indices[3] & (15 << (threadIdx.x * 4))) >> (threadIdx.x * 4);

            float temp = final_gmm[idx0] * final_gmm[idx1] - final_gmm[idx2] * final_gmm[idx3];

            if (final_gmm[10] > 0.0f)
            {
                final_gmm[4+threadIdx.x] = temp / final_gmm[10];
            }
            else
            {
                final_gmm[4+threadIdx.x] = 0.0f;
            }
        }

        if (threadIdx.x < 11)
        {
            gmm[blockIdx.x * gmm_pitch + threadIdx.x] = final_gmm[threadIdx.x];
        }
    }
}


// Single block, 32x2
__global__ void GMMcommonTerm(int gmmK, float *gmm, int gmm_pitch)
{
    __shared__ volatile float s_n[2][32];

    int gmm_idx = (threadIdx.x * 2) | threadIdx.y;

    float gmm_n = threadIdx.x < gmmK ? gmm[gmm_idx * gmm_pitch] : 0.0f;
    float sum = gmm_n;
    s_n[threadIdx.y][threadIdx.x] = sum;

    // Warp Reduction
    sum += s_n[threadIdx.y][(threadIdx.x + 16) & 31];
    s_n[threadIdx.y][threadIdx.x] = sum;

    sum += s_n[threadIdx.y][(threadIdx.x + 8) & 31];
    s_n[threadIdx.y][threadIdx.x] = sum;

    sum += s_n[threadIdx.y][(threadIdx.x + 4) & 31];
    s_n[threadIdx.y][threadIdx.x] = sum;

    sum += s_n[threadIdx.y][(threadIdx.x + 2) & 31];
    s_n[threadIdx.y][threadIdx.x] = sum;

    sum += s_n[threadIdx.y][(threadIdx.x + 1) & 31];

    if (threadIdx.x < gmmK)
    {
        float det = gmm[gmm_idx * gmm_pitch + 10];
        float commonTerm =  gmm_n / (sqrtf(det) * sum);

        gmm[gmm_idx * gmm_pitch + 10] = commonTerm;
    }
}

__device__ float GMMTerm(uchar4 pixel, const float *gmm)
{
    float3 v = make_float3(pixel.x - gmm[1], pixel.y - gmm[2], pixel.z - gmm[3]);

    float xxa = v.x * v.x * gmm[4];
    float yyd = v.y * v.y * gmm[7];
    float zzf = v.z * v.z * gmm[9];

    float yxb = v.x * v.y * gmm[5];
    float zxc = v.z * v.x * gmm[6];
    float zye = v.z * v.y * gmm[8];

    return gmm[10] * expf(-0.5f * (xxa + yyd + zzf + 2.0f * (yxb + zxc + zye)));
}

__global__ void GMMDataTermKernel(const uchar4 *image, int gmmN, const float *gmm, int gmm_pitch, float* output, int width, int height)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height) return;

    uchar4 pixel = image[x + y * width];

    float weights[MIXTURES];
    float weight_total = 0.0f;

    for(int i = 0; i < MIXTURES; i++)
    {
        float mixture_weight = 0.0f;

        for(int j = 0; j < gmmN; j += MIXTURES)
        {
            mixture_weight += GMMTerm(pixel, &gmm[(j + i) * gmm_pitch]);
        }

        weights[i] = mixture_weight;
        weight_total += mixture_weight;
    }

    for(int i = 0; i < MIXTURES; i++)
    {
        output[x + y * width + i * height * width] = weights[i] / weight_total;
    }
}

__device__
float3 normalize(float3 v)
{
    float norm = 1.0f / sqrtf(v.x * v.x + v.y * v.y + v.z * v.z);

    return make_float3(v.x * norm, v.y * norm, v.z * norm);
}

__device__
float3 mul_right(const float *M, float3 v)
{
    return make_float3(
               M[0] * v.x + M[1] * v.y + M[2] * v.z,
               M[1] * v.x + M[3] * v.y + M[4] * v.z,
               M[2] * v.x + M[4] * v.y + M[5] * v.z);
}

__device__
float largest_eigenvalue(const float *M)
{
    float norm = M[0] > M[3] ? M[0] : M[3];
    norm = M[0] > M[5] ? M[0] : M[5];
    norm = 1.0f / norm;

    float a00 = norm * M[0];
    float a01 = norm * M[1];
    float a02 = norm * M[2];
    float a11 = norm * M[3];
    float a12 = norm * M[4];
    float a22 = norm * M[5];

    float c0 = a00*a11*a22 + 2.0f*a01*a02*a12 - a00*a12*a12 - a11*a02*a02 - a22*a01*a01;
    float c1 = a00*a11 - a01*a01 + a00*a22 - a02*a02 + a11*a22 - a12*a12;
    float c2 = a00 + a11 + a22;

    const float inv3 = 1.0f / 3.0f;
    const float root3 = sqrtf(3.0f);

    float c2Div3 = c2*inv3;
    float aDiv3 = (c1 - c2*c2Div3)*inv3;

    if (aDiv3 > 0.0f)
    {
        aDiv3 = 0.0f;
    }

    float mbDiv2 = 0.5f*(c0 + c2Div3*(2.0f*c2Div3*c2Div3 - c1));
    float q = mbDiv2*mbDiv2 + aDiv3*aDiv3*aDiv3;

    if (q > 0.0f)
    {
        q = 0.0f;
    }

    float magnitude = sqrtf(-aDiv3);
    float angle = atan2(sqrtf(-q),mbDiv2)*inv3;
    float cs = cos(angle);
    float sn = sin(angle);

    float largest_eigenvalue = c2Div3 + 2.0f*magnitude*cs;

    float eigenvalue = c2Div3 - magnitude*(cs + root3*sn);

    if (eigenvalue > largest_eigenvalue)
    {
        largest_eigenvalue = eigenvalue;
    }

    eigenvalue = c2Div3 - magnitude*(cs - root3*sn);

    if (eigenvalue > largest_eigenvalue)
    {
        largest_eigenvalue = eigenvalue;
    }

    return largest_eigenvalue / norm;
}

__device__
float3 cross_prod(float3 a, float3 b)
{
    return make_float3((a.y*b.z)-(a.z*b.y), (a.z*b.x)-(a.x*b.z), (a.x*b.y)-(a.y*b.x));
}

__device__
float3 compute_eigenvector(const float *M, float eigenvalue)
{
    float3 r0 = make_float3(M[0] - eigenvalue, M[1], M[2]);
    float3 r1 = make_float3(M[2] , M[3]- eigenvalue, M[4]);

    float3 eigenvector = cross_prod(r0,r1);
    return normalize(eigenvector);
}

__device__
void largest_eigenvalue_eigenvector(const float *M, float3 &evec, float &eval)
{
    eval = largest_eigenvalue(M);
    evec = compute_eigenvector(M, eval);
}

__device__
float scalar_prod(float3 a, float3 b)
{
    return a.x * b.x + a.y * b.y + a.z * b.z;
}

struct GMMSplit_t
{
    int idx;
    float threshold;
    float3 eigenvector;
};

// 1 Block, 32x2
__global__ void GMMFindSplit(GMMSplit_t *gmmSplit, int gmmK, float *gmm, int gmm_pitch)
{
    __shared__ float s_eigenvalues[2][32];

    int gmm_idx = (threadIdx.x << 1) + threadIdx.y;

    float eigenvalue = 0;
    float3 eigenvector;

    if (threadIdx.x < gmmK)
    {
        largest_eigenvalue_eigenvector(&gmm[gmm_idx * gmm_pitch + 4], eigenvector, eigenvalue);
    }

    // Warp Reduction
    float maxvalue = eigenvalue;
    s_eigenvalues[threadIdx.y][threadIdx.x] = maxvalue;

    maxvalue = max(maxvalue, s_eigenvalues[threadIdx.y][(threadIdx.x+16) & 31]);
    s_eigenvalues[threadIdx.y][threadIdx.x] = maxvalue;

    maxvalue = max(maxvalue, s_eigenvalues[threadIdx.y][(threadIdx.x+8) & 31]);
    s_eigenvalues[threadIdx.y][threadIdx.x] = maxvalue;

    maxvalue = max(maxvalue, s_eigenvalues[threadIdx.y][(threadIdx.x+4) & 31]);
    s_eigenvalues[threadIdx.y][threadIdx.x] = maxvalue;

    maxvalue = max(maxvalue, s_eigenvalues[threadIdx.y][(threadIdx.x+2) & 31]);
    s_eigenvalues[threadIdx.y][threadIdx.x] = maxvalue;

    maxvalue = max(maxvalue, s_eigenvalues[threadIdx.y][(threadIdx.x+1) & 31]);

    if (maxvalue == eigenvalue)
    {
        GMMSplit_t split;

        split.idx = threadIdx.x;
        split.threshold = scalar_prod(make_float3(gmm[gmm_idx * gmm_pitch + 1], gmm[gmm_idx * gmm_pitch + 2], gmm[gmm_idx * gmm_pitch + 3]), eigenvector);
        split.eigenvector = eigenvector;

        gmmSplit[threadIdx.y] = split;
    }
}

__global__ void GMMDoSplit(const GMMSplit_t *gmmSplit, int k, float *gmm, int gmm_pitch, const uchar4 *image, char *alpha, int width, int height)
{
    __shared__ GMMSplit_t s_gmmSplit[2];

    int *s_linear = (int *) s_gmmSplit;
    int *g_linear = (int *) gmmSplit;

    if (threadIdx.y ==0 && threadIdx.x < 10)
    {
        s_linear[threadIdx.x] = g_linear[threadIdx.x];
    }

    __syncthreads();

    int x = blockIdx.x * 32 + threadIdx.x;
    int y0 = blockIdx.y * 32;

    for (int i = threadIdx.y; i < 32; i += blockDim.y)
    {
        int y = y0 + i;

        if (x < width && y < height)
        {
            char my_alpha = alpha[y * width + x];

            if(my_alpha != -1)
            {
                int select = my_alpha & 1;
                int gmm_idx = my_alpha >> 1;
    
                if (gmm_idx == s_gmmSplit[select].idx)
                {
                    // in the split cluster now
                    uchar4 pixel = image[y * width + x];
    
                    float value = scalar_prod(s_gmmSplit[select].eigenvector, make_float3(pixel.x, pixel.y, pixel.z));
    
                    if (value > s_gmmSplit[select].threshold)
                    {
                        // assign pixel to new cluster
                        alpha[y * width + x] =  k + select;
                    }
                }
            }
        }
    }
}

__global__ void InitializeImageAndAlphaKernel(float* input, int* labels, int width, int height, int channel_stride, uchar4* image, char* alpha)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height) return;

    int home = x + y * width;
    
    uchar4 color;
    color.x = input[home + 0 * channel_stride] * 255;
    color.y = input[home + 1 * channel_stride] * 255;
    color.z = input[home + 2 * channel_stride] * 255;

    image[home] = color;
    alpha[home] = labels[home];
}

#define BLOCK_SIZE 32
#define TILE(SIZE, STRIDE) (((SIZE - 1)/STRIDE) + 1)

void InitializeImageAndAlpha(float* input, int* labels, int width, int height, int channel_stride, uchar4* image, char* alpha)
{
    dim3 block_count = dim3(TILE(width, BLOCK_SIZE), TILE(height, BLOCK_SIZE));
    dim3 block_size = dim3(BLOCK_SIZE, BLOCK_SIZE);

    InitializeImageAndAlphaKernel<<<block_count, block_size>>>(input, labels, width, height, channel_stride, image, alpha);
}

void GMMInitialize(int gmm_N, float *gmm, float *scratch_mem, int gmm_pitch, const uchar4 *image, char *alpha, int width, int height)
{
    dim3 grid((width+31) / 32, (height+31) / 32);
    dim3 block(32,4);
    dim3 smallblock(32,2);

    for (int k = 2; k < gmm_N; k+=2)
    {
        GMMReductionKernel<4, true><<<grid, block>>>(0, &scratch_mem[grid.x *grid.y], gmm_pitch/4, image, alpha, width, height, (unsigned int *) scratch_mem);

        for (int i=1; i < k; ++i)
        {
            GMMReductionKernel<4, false><<<grid, block>>>(i, &scratch_mem[grid.x *grid.y], gmm_pitch/4, image, alpha, width, height, (unsigned int *) scratch_mem);
        }

        GMMFinalizeKernel<4, false><<<k, 32 *4>>>(gmm, &scratch_mem[grid.x *grid.y], gmm_pitch/4, grid.x *grid.y);

        GMMFindSplit<<<1, smallblock>>>((GMMSplit_t *) scratch_mem, k / 2, gmm, gmm_pitch/4);

        GMMDoSplit<<<grid, block>>>((GMMSplit_t *) scratch_mem, (k/2) << 1, gmm, gmm_pitch/4, image, alpha, width, height);
    }
}

void GMMUpdate(int gmm_N, float *gmm, float *scratch_mem, int gmm_pitch, const uchar4 *image, char *alpha, int width, int height)
{
    dim3 grid((width+31) / 32, (height+31) / 32);
    dim3 block(32,4);

    GMMReductionKernel<4, true><<<grid, block>>>(0, &scratch_mem[grid.x *grid.y], gmm_pitch/4, image, alpha, width, height, (unsigned int *) scratch_mem);

    for (int i=1; i<gmm_N; ++i)
    {
        GMMReductionKernel<4, false><<<grid, block>>>(i, &scratch_mem[grid.x *grid.y], gmm_pitch/4, image, alpha, width, height, (unsigned int *) scratch_mem);
    }

    GMMFinalizeKernel<4, true><<<gmm_N, 32 *4>>>(gmm, &scratch_mem[grid.x *grid.y], gmm_pitch/4, grid.x *grid.y);

    block.x = 32;
    block.y = 2;
    GMMcommonTerm<<<1, block>>>(gmm_N / 2, gmm, gmm_pitch/4);
}

void GMMDataTerm(const uchar4 *image, int gmmN, const float *gmm, int gmm_pitch, float* output, int width, int height)
{
    dim3 block(32,8);
    dim3 grid((width+block.x-1) / block.x, (height+block.y-1) / block.y);

    GMMDataTermKernel<<<grid, block>>>(image, gmmN, gmm, gmm_pitch/4, output, width, height);
}
