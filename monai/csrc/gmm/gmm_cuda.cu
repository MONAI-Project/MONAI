
#include <hip/hip_runtime.h>
/*
Copyright 2020 - 2021 MONAI Consortium
Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at
    http://www.apache.org/licenses/LICENSE-2.0
Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/

#define BLOCK_SIZE 32
#define TILE(SIZE, STRIDE) ((((SIZE) - 1)/(STRIDE)) + 1)

#define CHANNELS 3
#define MAX_CHANNELS 16
#define MAX_MIXTURES 16

__device__ __forceinline__ float get_component(float* pixel, int i)
{
    switch (i)
    {
        case 0 : return 1.0f;
        case 1 : return pixel[0];
        case 2 : return pixel[1];
        case 3 : return pixel[2];
        case 4 : return pixel[0] * pixel[0];
        case 5 : return pixel[0] * pixel[1];
        case 6 : return pixel[0] * pixel[2];
        case 7 : return pixel[1] * pixel[1];
        case 8 : return pixel[1] * pixel[2];
        case 9 : return pixel[2] * pixel[2];
    };

    return 0.0f;
}

__device__ __forceinline__ float get_constant(float *gmm, int i)
{
    const float epsilon = 1.0e-3f;

    switch (i)
    {
        case 0 : return 0.0f;
        case 1 : return 0.0f;
        case 2 : return 0.0f;
        case 3 : return 0.0f;
        case 4 : return gmm[1] * gmm[1] + epsilon;
        case 5 : return gmm[1] * gmm[2];
        case 6 : return gmm[1] * gmm[3];
        case 7 : return gmm[2] * gmm[2] + epsilon;
        case 8 : return gmm[2] * gmm[3];
        case 9 : return gmm[3] * gmm[3] + epsilon;
    };

    return 0.0f;
}


// Tile Size: 32x32, Block Size 32xwarp_N
template<int warp_N, bool create_gmm_flags>
__global__ void GMMReductionKernel(int gmm_idx, const float* image, const int* alpha, float* gmm, unsigned int* tile_gmms, int element_count, int component_count, int mixture_count)
{
    __shared__ float s_lists[32 * 32 * CHANNELS];
    __shared__ volatile float s_gmm[32 * warp_N];
    __shared__ float s_final[warp_N];

    __shared__ int gmm_flags[32];

    int warp_idx = threadIdx.x >> 5;
    int thread_idx = threadIdx.x;
    int lane_idx = threadIdx.x & 31;

    int block_idx = blockIdx.x;

    float *block_gmm = &gmm[(gridDim.x * gmm_idx + block_idx) * component_count];
    volatile float *warp_gmm = &s_gmm[warp_idx * 32];

    if (create_gmm_flags)
    {
        if (warp_idx == 0)
        {
            gmm_flags[lane_idx] = 0;
        }

        __syncthreads();
    }
    else
    {
        unsigned int gmm_mask = tile_gmms[block_idx];

        if ((gmm_mask & (1u << gmm_idx)) == 0)
        {

            if (lane_idx < 10 && warp_idx == 0)
            {
                block_gmm[lane_idx] = 0.0f;
            }

            return;
        }
    }

    int list_idx = 0;

    int index = block_idx * 32 * 32 + warp_idx * 32 + lane_idx;

    // Build lists of pixels that belong to this GMM

    for (int k=0; k < (32/warp_N); ++k)
    {
        if (index < element_count)
        {
            int my_alpha = alpha[index];

            if (my_alpha != -1)
            {
                int my_gmm_idx = (my_alpha & 15) + (my_alpha >> 4) * mixture_count;

                if (create_gmm_flags)
                {
                    gmm_flags[my_gmm_idx] = 1;
                }
    
                if (my_gmm_idx == gmm_idx)
                {
                    s_lists[(thread_idx + list_idx * (32*warp_N)) * CHANNELS + 0] = image[index + 0 * element_count] * 255;
                    s_lists[(thread_idx + list_idx * (32*warp_N)) * CHANNELS + 1] = image[index + 1 * element_count] * 255;
                    s_lists[(thread_idx + list_idx * (32*warp_N)) * CHANNELS + 2] = image[index + 2 * element_count] * 255;
                    ++list_idx;
                }
            }
        }

        index += warp_N * 32;
    }

    __syncthreads();

    if (warp_idx == 0 && create_gmm_flags)
    {
        tile_gmms[block_idx] = __ballot_sync(0xFFFFFFFF, gmm_flags[lane_idx] > 0);
    }

    // Reduce for each global GMM element

    for (int i=0; i<10; ++i)
    {
        float thread_gmm;

        if (i == 0)
        {
            // thread_gmm = list_idx for first component
            thread_gmm = list_idx;
        }
        else
        {
            float temp_array[CHANNELS];
            temp_array[0] = s_lists[thread_idx * CHANNELS + 0];
            temp_array[1] = s_lists[thread_idx * CHANNELS + 1];
            temp_array[2] = s_lists[thread_idx * CHANNELS + 2];

            thread_gmm = list_idx > 0 ? get_component(temp_array, i) : 0.0f;

            for (int k=1; k<(32/warp_N) && k < list_idx; ++k)
            {
                temp_array[0] = s_lists[(thread_idx + k * (32*warp_N)) * CHANNELS + 0];
                temp_array[1] = s_lists[(thread_idx + k * (32*warp_N)) * CHANNELS + 1];
                temp_array[2] = s_lists[(thread_idx + k * (32*warp_N)) * CHANNELS + 2];

                thread_gmm += get_component(temp_array, i);
            }
        }

        warp_gmm[lane_idx] = thread_gmm;

        // Warp Reductions
        thread_gmm += warp_gmm[(lane_idx + 16) & 31];
        warp_gmm[lane_idx] = thread_gmm;

        thread_gmm += warp_gmm[(lane_idx + 8) & 31];
        warp_gmm[lane_idx] = thread_gmm;

        thread_gmm += warp_gmm[(lane_idx + 4) & 31];
        warp_gmm[lane_idx] = thread_gmm;

        thread_gmm += warp_gmm[(lane_idx + 2) & 31];
        warp_gmm[lane_idx] = thread_gmm;

        thread_gmm += warp_gmm[(lane_idx + 1) & 31];
        s_final[warp_idx] = thread_gmm;

        __syncthreads();

        // Final Reduction
        if (warp_idx == 0 && lane_idx == 0)
        {
            for (int j=1; j<warp_N; ++j)
            {
                thread_gmm += s_final[j];
            }

            block_gmm[i] = thread_gmm;
        }
    }
}

__constant__ int det_indices[] = { (9 << (4*4)) + (4 << (3*4)) + (6 << (2*4)) + (5 << (1*4)) + (4 << (0*4)),
                                   (5 << (4*4)) + (8 << (3*4)) + (6 << (2*4)) + (6 << (1*4)) + (7 << (0*4)),
                                   (5 << (4*4)) + (8 << (3*4)) + (7 << (2*4)) + (8 << (1*4)) + (9 << (0*4))
                                 };

__constant__ int inv_indices[] = { (4 << (5*4)) + (5 << (4*4)) + (4 << (3*4)) + (5 << (2*4)) + (6 << (1*4)) + (7 << (0*4)),
                                   (7 << (5*4)) + (6 << (4*4)) + (9 << (3*4)) + (8 << (2*4)) + (8 << (1*4)) + (9 << (0*4)),
                                   (5 << (5*4)) + (4 << (4*4)) + (6 << (3*4)) + (6 << (2*4)) + (5 << (1*4)) + (8 << (0*4)),
                                   (5 << (5*4)) + (8 << (4*4)) + (6 << (3*4)) + (7 << (2*4)) + (9 << (1*4)) + (8 << (0*4))
                                 };


// One block per GMM, 32*warp_N threads (1-dim)
template <int warp_N, bool invertSigma>
__global__ void GMMFinalizeKernel(const float *gmm_scratch, float *gmm, int gmm_stride, int component_count)
{
    __shared__ volatile float s_gmm[warp_N*32];
    __shared__ float s_final[warp_N];
    __shared__ float final_gmm[15];

    const int thread_N = warp_N * 32;

    const float *gmm_partial = &gmm_scratch[blockIdx.x * gmm_stride * component_count];

    volatile float *warp_gmm = &s_gmm[threadIdx.x & 0x0ffe0];

    int thread_idx = threadIdx.x;
    int lane_idx = threadIdx.x & 31;
    int warp_idx = threadIdx.x >> 5;

    float norm_factor = 1.0f;

    for (int i=0; i<10; ++i)
    {
        float thread_gmm = 0.0f;

        for (int j=thread_idx; j < gmm_stride; j+= thread_N)
        {
            thread_gmm += gmm_partial[j * component_count + i];
        }

        warp_gmm[lane_idx] = thread_gmm;

        // Warp Reduction
        thread_gmm += warp_gmm[(lane_idx + 16) & 31];
        warp_gmm[lane_idx] = thread_gmm;

        thread_gmm += warp_gmm[(lane_idx + 8) & 31];
        warp_gmm[lane_idx] = thread_gmm;

        thread_gmm += warp_gmm[(lane_idx + 4) & 31];
        warp_gmm[lane_idx] = thread_gmm;

        thread_gmm += warp_gmm[(lane_idx + 2) & 31];
        warp_gmm[lane_idx] = thread_gmm;

        thread_gmm += warp_gmm[(lane_idx + 1) & 31];

        s_final[warp_idx] = thread_gmm;

        __syncthreads();

        // Final Reduction
        if (warp_idx == 0 && lane_idx == 0)
        {
            for (int j=1; j<warp_N; ++j)
            {
                thread_gmm += s_final[j];
            }

            final_gmm[i] = norm_factor * thread_gmm - get_constant(final_gmm, i);

            if (i == 0)
            {
                if (thread_gmm > 0)
                {
                    norm_factor = 1.0f / thread_gmm;
                }
            }
        }
    }

    if (threadIdx.y == 0)
    {
        // Compute det(Sigma) using final_gmm [10-14] as scratch mem

        if (threadIdx.x < 5)
        {

            int idx0 = (det_indices[0] & (15 << (threadIdx.x * 4))) >> (threadIdx.x * 4);
            int idx1 = (det_indices[1] & (15 << (threadIdx.x * 4))) >> (threadIdx.x * 4);
            int idx2 = (det_indices[2] & (15 << (threadIdx.x * 4))) >> (threadIdx.x * 4);

            final_gmm[10 + threadIdx.x] = final_gmm[idx0] * final_gmm[idx1] * final_gmm[idx2];

            float det = final_gmm[10] + 2.0f * final_gmm[11] - final_gmm[12] - final_gmm[13] - final_gmm[14];
            final_gmm[10] = det;
        }

        // Compute inv(Sigma)
        if (invertSigma && threadIdx.x < 6)
        {
            int idx0 = (inv_indices[0] & (15 << (threadIdx.x * 4))) >> (threadIdx.x * 4);
            int idx1 = (inv_indices[1] & (15 << (threadIdx.x * 4))) >> (threadIdx.x * 4);
            int idx2 = (inv_indices[2] & (15 << (threadIdx.x * 4))) >> (threadIdx.x * 4);
            int idx3 = (inv_indices[3] & (15 << (threadIdx.x * 4))) >> (threadIdx.x * 4);

            float temp = final_gmm[idx0] * final_gmm[idx1] - final_gmm[idx2] * final_gmm[idx3];

            if (final_gmm[10] > 0.0f)
            {
                final_gmm[4+threadIdx.x] = temp / final_gmm[10];
            }
            else
            {
                final_gmm[4+threadIdx.x] = 0.0f;
            }
        }

        if (threadIdx.x < 11)
        {
            gmm[blockIdx.x * component_count + threadIdx.x] = final_gmm[threadIdx.x];
        }
    }
}


// Single block, 32x2
__global__ void GMMcommonTerm(float *gmm, int mixture_count, int mixture_size, int component_count)
{
    __shared__ volatile float s_n[MAX_MIXTURES][32];

    int gmm_idx = (threadIdx.x * mixture_count) | threadIdx.y;

    float gmm_n = threadIdx.x < mixture_size ? gmm[gmm_idx * component_count] : 0.0f;
    float sum = gmm_n;
    s_n[threadIdx.y][threadIdx.x] = sum;

    // Warp Reduction
    sum += s_n[threadIdx.y][(threadIdx.x + 16) & 31];
    s_n[threadIdx.y][threadIdx.x] = sum;

    sum += s_n[threadIdx.y][(threadIdx.x + 8) & 31];
    s_n[threadIdx.y][threadIdx.x] = sum;

    sum += s_n[threadIdx.y][(threadIdx.x + 4) & 31];
    s_n[threadIdx.y][threadIdx.x] = sum;

    sum += s_n[threadIdx.y][(threadIdx.x + 2) & 31];
    s_n[threadIdx.y][threadIdx.x] = sum;

    sum += s_n[threadIdx.y][(threadIdx.x + 1) & 31];

    if (threadIdx.x < mixture_size)
    {
        float det = gmm[gmm_idx * component_count + 10];
        float commonTerm =  gmm_n / (sqrtf(det) * sum);

        gmm[gmm_idx * component_count + 10] = commonTerm;
    }
}

__device__ float GMMTerm(float* pixel, const float *gmm)
{
    float3 v = make_float3(pixel[0] - gmm[1], pixel[1] - gmm[2], pixel[2] - gmm[3]);

    float xxa = v.x * v.x * gmm[4];
    float yyd = v.y * v.y * gmm[7];
    float zzf = v.z * v.z * gmm[9];

    float yxb = v.x * v.y * gmm[5];
    float zxc = v.z * v.x * gmm[6];
    float zye = v.z * v.y * gmm[8];

    return gmm[10] * expf(-0.5f * (xxa + yyd + zzf + 2.0f * (yxb + zxc + zye)));
}

__global__ void GMMDataTermKernel(const float *image, const float *gmm, float* output, int element_count, int mixture_count, int mixture_size, int component_count)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index >= element_count) return;

    float temp_array[CHANNELS];
    temp_array[0] = image[index + 0 * element_count] * 255;
    temp_array[1] = image[index + 1 * element_count] * 255;
    temp_array[2] = image[index + 2 * element_count] * 255;

    float weights[MAX_MIXTURES];
    float weight_total = 0.0f;

    for(int i = 0; i < mixture_count; i++)
    {
        float mixture_weight = 0.0f;

        for(int j = 0; j < mixture_size; j++)
        {
            mixture_weight += GMMTerm(temp_array, &gmm[(mixture_count * j + i) * component_count]);
        }

        weights[i] = mixture_weight;
        weight_total += mixture_weight;
    }

    for(int i = 0; i < mixture_count; i++)
    {
        output[index + i * element_count] = weights[i] / weight_total;
    }
}

__device__
float3 normalize(float3 v)
{
    float norm = 1.0f / sqrtf(v.x * v.x + v.y * v.y + v.z * v.z);

    return make_float3(v.x * norm, v.y * norm, v.z * norm);
}

__device__
float3 mul_right(const float *M, float3 v)
{
    return make_float3(
               M[0] * v.x + M[1] * v.y + M[2] * v.z,
               M[1] * v.x + M[3] * v.y + M[4] * v.z,
               M[2] * v.x + M[4] * v.y + M[5] * v.z);
}

__device__
float largest_eigenvalue(const float *M)
{
    float norm = M[0] > M[3] ? M[0] : M[3];
    norm = M[0] > M[5] ? M[0] : M[5];
    norm = 1.0f / norm;

    float a00 = norm * M[0];
    float a01 = norm * M[1];
    float a02 = norm * M[2];
    float a11 = norm * M[3];
    float a12 = norm * M[4];
    float a22 = norm * M[5];

    float c0 = a00*a11*a22 + 2.0f*a01*a02*a12 - a00*a12*a12 - a11*a02*a02 - a22*a01*a01;
    float c1 = a00*a11 - a01*a01 + a00*a22 - a02*a02 + a11*a22 - a12*a12;
    float c2 = a00 + a11 + a22;

    const float inv3 = 1.0f / 3.0f;
    const float root3 = sqrtf(3.0f);

    float c2Div3 = c2*inv3;
    float aDiv3 = (c1 - c2*c2Div3)*inv3;

    if (aDiv3 > 0.0f)
    {
        aDiv3 = 0.0f;
    }

    float mbDiv2 = 0.5f*(c0 + c2Div3*(2.0f*c2Div3*c2Div3 - c1));
    float q = mbDiv2*mbDiv2 + aDiv3*aDiv3*aDiv3;

    if (q > 0.0f)
    {
        q = 0.0f;
    }

    float magnitude = sqrtf(-aDiv3);
    float angle = atan2(sqrtf(-q),mbDiv2)*inv3;
    float cs = cos(angle);
    float sn = sin(angle);

    float largest_eigenvalue = c2Div3 + 2.0f*magnitude*cs;

    float eigenvalue = c2Div3 - magnitude*(cs + root3*sn);

    if (eigenvalue > largest_eigenvalue)
    {
        largest_eigenvalue = eigenvalue;
    }

    eigenvalue = c2Div3 - magnitude*(cs - root3*sn);

    if (eigenvalue > largest_eigenvalue)
    {
        largest_eigenvalue = eigenvalue;
    }

    return largest_eigenvalue / norm;
}

__device__
float3 cross_prod(float3 a, float3 b)
{
    return make_float3((a.y*b.z)-(a.z*b.y), (a.z*b.x)-(a.x*b.z), (a.x*b.y)-(a.y*b.x));
}

__device__
float3 compute_eigenvector(const float *M, float eigenvalue)
{
    float3 r0 = make_float3(M[0] - eigenvalue, M[1], M[2]);
    float3 r1 = make_float3(M[2] , M[3]- eigenvalue, M[4]);

    float3 eigenvector = cross_prod(r0,r1);
    return normalize(eigenvector);
}

__device__
void largest_eigenvalue_eigenvector(const float *M, float3 &evec, float &eval)
{
    eval = largest_eigenvalue(M);
    evec = compute_eigenvector(M, eval);
}

__device__
float scalar_prod(float3 a, float3 b)
{
    return a.x * b.x + a.y * b.y + a.z * b.z;
}

struct GMMSplit_t
{
    int idx;
    float threshold;
    float3 eigenvector;
};

// 1 Block, 32xmixture_count
__global__ void GMMFindSplit(GMMSplit_t *gmmSplit, int gmmK, float *gmm, int component_count, int mixture_count)
{
    __shared__ float s_eigenvalues[MAX_MIXTURES][32];

    // int gmm_idx = (threadIdx.x << 1) + threadIdx.y;
    int gmm_idx = threadIdx.x * mixture_count + threadIdx.y;

    float eigenvalue = 0;
    float3 eigenvector;

    if (threadIdx.x < gmmK)
    {
        largest_eigenvalue_eigenvector(&gmm[gmm_idx * component_count + 4], eigenvector, eigenvalue);
    }

    // Warp Reduction
    float maxvalue = eigenvalue;
    s_eigenvalues[threadIdx.y][threadIdx.x] = maxvalue;

    maxvalue = max(maxvalue, s_eigenvalues[threadIdx.y][(threadIdx.x+16) & 31]);
    s_eigenvalues[threadIdx.y][threadIdx.x] = maxvalue;

    maxvalue = max(maxvalue, s_eigenvalues[threadIdx.y][(threadIdx.x+8) & 31]);
    s_eigenvalues[threadIdx.y][threadIdx.x] = maxvalue;

    maxvalue = max(maxvalue, s_eigenvalues[threadIdx.y][(threadIdx.x+4) & 31]);
    s_eigenvalues[threadIdx.y][threadIdx.x] = maxvalue;

    maxvalue = max(maxvalue, s_eigenvalues[threadIdx.y][(threadIdx.x+2) & 31]);
    s_eigenvalues[threadIdx.y][threadIdx.x] = maxvalue;

    maxvalue = max(maxvalue, s_eigenvalues[threadIdx.y][(threadIdx.x+1) & 31]);

    if (maxvalue == eigenvalue)
    {
        GMMSplit_t split;

        split.idx = threadIdx.x;
        split.threshold = scalar_prod(make_float3(gmm[gmm_idx * component_count + 1], gmm[gmm_idx * component_count + 2], gmm[gmm_idx * component_count + 3]), eigenvector);
        split.eigenvector = eigenvector;

        gmmSplit[threadIdx.y] = split;
    }
}

#define DO_SPLIT_DEGENERACY 4

__global__ void GMMDoSplit(const GMMSplit_t *gmmSplit, int k, float *gmm, int component_count, const float *image, int *alpha, int element_count)
{
    __shared__ GMMSplit_t s_gmmSplit[MAX_MIXTURES];

    int *s_linear = (int *) s_gmmSplit;
    int *g_linear = (int *) gmmSplit;

    if (threadIdx.x < 10)
    {
        s_linear[threadIdx.x] = g_linear[threadIdx.x];
    }

    __syncthreads();

    int index = threadIdx.x + blockIdx.x * BLOCK_SIZE * DO_SPLIT_DEGENERACY;

    for (int i = 0; i < DO_SPLIT_DEGENERACY; i++)
    {
        index += BLOCK_SIZE;

        if (index < element_count)
        {
            int my_alpha = alpha[index];

            if(my_alpha != -1)
            {
                int select = my_alpha & 15;
                int gmm_idx = my_alpha >> 4;
    
                if (gmm_idx == s_gmmSplit[select].idx)
                {
                    // in the split cluster now
                    float temp_array[CHANNELS];
                    temp_array[0] = image[index + 0 * element_count] * 255;
                    temp_array[1] = image[index + 1 * element_count] * 255;
                    temp_array[2] = image[index + 2 * element_count] * 255;
    
                    float value = scalar_prod(s_gmmSplit[select].eigenvector, make_float3(temp_array[0], temp_array[1], temp_array[2]));
    
                    if (value > s_gmmSplit[select].threshold)
                    {
                        // assign pixel to new cluster
                        alpha[index] =  k + select;
                    }
                }
            }
        }
    }
}

void GMMInitialize(const float *image, int *alpha, float *gmm, float *scratch_mem, int element_count, int mixture_count, int mixture_size, int component_count)
{
    dim3 grid(TILE(element_count, BLOCK_SIZE * BLOCK_SIZE));
    dim3 block(BLOCK_SIZE * 4);
    
    float* block_gmm_scratch = &scratch_mem[grid.x];
    unsigned int* block_active_scratch = (unsigned int*)scratch_mem;
    GMMSplit_t* gmm_split_scratch = (GMMSplit_t*) scratch_mem;

    int gmm_N = mixture_count * mixture_size;

    for (int k = mixture_count; k < gmm_N; k+=mixture_count)
    {
        GMMReductionKernel<4, true><<<grid, block>>>(0, image, alpha, block_gmm_scratch, block_active_scratch, element_count, component_count, mixture_count);

        for (int i=1; i < k; ++i)
        {
            GMMReductionKernel<4, false><<<grid, block>>>(i, image, alpha, block_gmm_scratch, block_active_scratch, element_count, component_count, mixture_count);
        }

        GMMFinalizeKernel<4, false><<<k, block>>>(block_gmm_scratch, gmm, grid.x, component_count);

        GMMFindSplit<<<1, dim3(BLOCK_SIZE, mixture_count)>>>(gmm_split_scratch, k / mixture_count, gmm, component_count, mixture_count);
        GMMDoSplit<<<TILE(element_count, BLOCK_SIZE * DO_SPLIT_DEGENERACY), BLOCK_SIZE>>>(gmm_split_scratch, (k / mixture_count) << 4, gmm, component_count, image, alpha, element_count);
    }
}

void GMMUpdate(const float *image, int *alpha, float *gmm, float *scratch_mem, int element_count, int mixture_count, int mixture_size, int component_count)
{
    dim3 grid(TILE(element_count, BLOCK_SIZE * BLOCK_SIZE));
    dim3 block(BLOCK_SIZE * 4);

    float* block_gmm_scratch = &scratch_mem[grid.x];
    unsigned int* block_active_scratch = (unsigned int*)scratch_mem;

    int gmm_N = mixture_count * mixture_size;

    GMMReductionKernel<4, true><<<grid, block>>>(0, image, alpha, block_gmm_scratch, block_active_scratch, element_count, component_count, mixture_count);

    for (int i = 1; i < gmm_N; ++i)
    {
        GMMReductionKernel<4, false><<<grid, block>>>(i, image, alpha, block_gmm_scratch, block_active_scratch, element_count, component_count, mixture_count);
    }

    GMMFinalizeKernel<4, true><<<gmm_N, block>>>(block_gmm_scratch, gmm, grid.x, component_count);

    GMMcommonTerm<<<1, dim3(BLOCK_SIZE, mixture_count)>>>(gmm, mixture_count, mixture_size, component_count);
}

void GMMDataTerm(const float *image, const float *gmm, float* output, int element_count, int mixture_count, int mixture_size, int component_count)
{
    dim3 block(BLOCK_SIZE, 1);
    dim3 grid(TILE(element_count, BLOCK_SIZE), 1);

    GMMDataTermKernel<<<grid, block>>>(image, gmm, output, element_count, mixture_count, mixture_size, component_count);
}

void GMM_Cuda(const float* input, const int* labels, float* output, int batch_count, int channel_count, int element_count, int mixture_count, int mixture_size)
{
    int component_count = 1 + (channel_count + 1) * (channel_count + 2) / 2;
    int gmm_size = component_count * mixture_count * mixture_size;

    float* scratch_mem = output;
    float* gmm; 
    int* alpha;

    hipMalloc(&gmm, gmm_size * sizeof(float));
    hipMalloc(&alpha, element_count * sizeof(int));

    hipMemcpyAsync(alpha, labels, element_count * sizeof(int), hipMemcpyDeviceToDevice);
    
    GMMInitialize(input, alpha, gmm, scratch_mem, element_count, mixture_count, mixture_size, component_count);
    GMMUpdate(input, alpha, gmm, scratch_mem, element_count, mixture_count, mixture_size, component_count);
    GMMDataTerm(input, gmm, output, element_count, mixture_count, mixture_size, component_count);

    hipFree(alpha);
    hipFree(gmm);
}
